#include "hip/hip_runtime.h"
/*
 * RaceTrap implementation based on RaceTrap.java
 *
 * Created on 22. juni 2000, 13:48
 * 
 * Brian Vinter
 * 
 * Modified by John Markus Bjørndalen, 2008-12-04, 2009-10-15. 
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include "graphicsScreen.h"
#include "StopWatch.h"
#include "stack.c"
#include "deviceStack.c"
#include <pthread.h>
#include <hip/hip_runtime.h>


typedef struct {
  int x; 
  int y; 
} Coord; 

typedef struct {
  float        length;     // Length of the current path (distance)
  unsigned char nCitiesVisited;    // Number of bags currently placed
  unsigned char path[0];    // Array of vertex/bag numbers in the path (see comment in
  // Alloc_RouteDefinition())
} RouteDefinition; 


pthread_mutex_t mut = PTHREAD_MUTEX_INITIALIZER;


int      nTotalCities = 0;             // Number of grain-bagsg
Coord   *cityCoords;             // Coordinates for the grain-bags
float **distanceTable;         // Table of distances between any two grain-bags
float   maxRouteLen = 10E100;  // Initial best distance, must be longer than any possible route
float   globalBest  = 10E100;  // Bounding variable

int fanOutLevel = 6;
int elemSize = 0;
int arraySize = 0;
int dist_array_size = 0;


inline RouteDefinition* Alloc_RouteDefinition()
{
  if (nTotalCities <= 0) 
  {
    fprintf(stderr, "Error: Alloc_RouteDefinition called with invalid nTotalCities (%d)\n", nTotalCities); 
    exit(-1); 
  }
  // NB: The +nTotalCities*sizeof.. trick "expands" the path[0] array in RouteDefintion
  // to a path[nTotalCities] array.
  RouteDefinition *def = NULL;  
  return (RouteDefinition*) malloc(sizeof(RouteDefinition) + nTotalCities * sizeof(def->path[0]));
}

__device__ RouteDefinition* device_alloc_RouteDefinition(int nTotCities)
{
  if (nTotCities <= 0) 
  {
    printf("Error: Alloc_RouteDefinition called with invalid nTotalCities (%d)\n", nTotCities); 
    //exit(-1); 
  }
  // NB: The +nTotalCities*sizeof.. trick "expands" the path[0] array in RouteDefintion
  // to a path[nTotalCities] array.
  RouteDefinition *def = NULL;  
  return (RouteDefinition*) malloc(sizeof(RouteDefinition) + nTotCities * sizeof(def->path[0]));
}



#ifdef GRAPHICS
// Plots a route on the display
void PlotRoute(char *path)
{ 
  int i;
  gs_clear(WHITE);
  // Plot each grain bag
  for(i = 0; i < nTotalCities; i++)
    gs_dot(cityCoords[i].x, cityCoords[i].y, 10, RED); 
  
  // Plot edges in the path
    for(i = 0; i < nTotalCities-1; i++)
      gs_line(cityCoords[(int)path[i  ]].x, cityCoords[(int)path[i  ]].y, 
	      cityCoords[(int)path[i+1]].x, cityCoords[(int)path[i+1]].y, RED);    
      
      // Plot the final edge closing the path
      gs_line(cityCoords[(int)path[nTotalCities-1]].x, cityCoords[(int)path[nTotalCities-1]].y, 
	      cityCoords[(int)path[0      ]].x, cityCoords[(int)path[0      ]].y, RED);
      
      gs_update();
}
#endif

// Function for making the 2d array of distances into a linear array for the device
float *flatten_dist_table()
{
  dist_array_size  = sizeof(float) * (nTotalCities * nTotalCities); 
  float* dist_array = (float*)malloc(dist_array_size);
  
  float *iter = dist_array;
  for (int a = 0; a < nTotalCities; a++){
    for (int b = 0; b < nTotalCities; b++){
      iter[a*nTotalCities+b] = distanceTable[a][b];
    }
  }
    
  return dist_array;  
}

// calculates the number of nodes at a given level in the three
int nodesAtLevel(int level)
{ 
  int result = 1;//nTotalCities;
  int tmp = 1;   
  for (int i = 1; i < level; i++){
    
    result += (nTotalCities - i)*(tmp);
    tmp = (nTotalCities - i)*(tmp);

  }  
  return tmp;
}

// Helper function for printing out a route
void print_route(RouteDefinition *route)
{
    printf("Route - nVisited: %d, nLen: %f, route: ", route->nCitiesVisited, route->length);
    for (int c = 0; c < nTotalCities; c++){
       printf("%d-", route->path[c]);
    }
    printf("\n");

}


// Make a stack into an array for device
char* stackToArray(stack_t *stck)
{  
  RouteDefinition *def = NULL;
  elemSize = sizeof(RouteDefinition) + nTotalCities * sizeof(def->path[0]);
  
  int rest = elemSize % 4;
  int padding = 0; 
  
  if (rest > 0){
    padding = 4 - rest;
  }
  
  elemSize = elemSize + padding;
  arraySize = elemSize * nodesAtLevel(fanOutLevel);

  char *tmp_route, *array;//, *iter;
  array = (char*)malloc(arraySize);
   
  for (int p = 0; p < arraySize; p += elemSize) {
    tmp_route = (char*)pop_back(stck); 
    
    memcpy(array + p, tmp_route, elemSize - padding);

    free(tmp_route);
  }
  return array;
}

// Helper function for calculatint dist between cities
__device__ float calcDist(float *distTable, int a, int b, int nTotCities)
{
  return distTable[a*nTotCities+b];
}

// fincds the best route in its subtree
__device__ RouteDefinition* findBestRoute(RouteDefinition *route, float *devDistArray, int nTotCities, int routeSize)
{
  stack_t* stck = device_stack_create();
  
  RouteDefinition *bestRoute;
  bestRoute = device_alloc_RouteDefinition(nTotCities);
  bestRoute->length = 10E100;
  
  RouteDefinition *curr_route;
  curr_route = device_alloc_RouteDefinition(nTotCities);
  
  RouteDefinition *newRoute;
  
  float newLength;
  
  memcpy(curr_route, route, routeSize);
  
  device_push(stck, curr_route);
  
  while(stck->size > 0){
    
    curr_route = (RouteDefinition*)device_pop(stck);
    
    if (curr_route->nCitiesVisited == nTotCities){
      
      curr_route->length += calcDist(devDistArray, curr_route->path[curr_route->nCitiesVisited-1], curr_route->path[0], nTotCities);
      
      if (curr_route->length < bestRoute->length){
	free(bestRoute);
	bestRoute = curr_route;
      } else {
	free(curr_route);
      }
      
    } else {
      for (int i = curr_route->nCitiesVisited; i < nTotCities; i++){
	newLength = curr_route->length + calcDist(devDistArray, curr_route->path[curr_route->nCitiesVisited-1], curr_route->path[i], nTotCities);
	
	if (newLength >= bestRoute->length){     
	  continue;
	}
	
	newRoute = device_alloc_RouteDefinition(nTotCities);
	
	memcpy(newRoute->path, curr_route->path, nTotCities);
	
	newRoute->path[curr_route->nCitiesVisited] = curr_route->path[i];
	newRoute->path[i]              = curr_route->path[curr_route->nCitiesVisited]; 
	newRoute->nCitiesVisited = curr_route->nCitiesVisited + 1;
	newRoute->length  = newLength;
		    
	device_push(stck, newRoute);
	
      }
      free(curr_route);
    }
    
  }
  
  device_stack_destroy(stck);
  
  memcpy(route, bestRoute, routeSize);
  
  return route;
}


__global__ void cudaSolve(char* array, float *devDistArray, int numRoutes, int routeSize, int numCities)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  
  if (i < numRoutes){
    array = array + (i * routeSize);
    
    RouteDefinition *route = (RouteDefinition*)array;
    
    findBestRoute(route, devDistArray, numCities, routeSize);
  }
}


RouteDefinition* findBestRouteInArray(char *array){
  
  RouteDefinition *tmp_route, *bestRoute;

  bestRoute = Alloc_RouteDefinition();
  bestRoute->length = maxRouteLen;
  
  for (int p = 0; p < arraySize; p += elemSize) {
    
    tmp_route = (RouteDefinition*)(array + p);
    
    //print_route(tmp_route);
    
    if (tmp_route->length < bestRoute->length){
      //printf("NEW BEST\n");
      bestRoute = tmp_route;
#ifdef GRAPHICS
      PlotRoute((char *)bestRoute->path);
      //sleep(1);
#endif  
    } 
  }
  
  tmp_route = Alloc_RouteDefinition();
  
  memcpy(tmp_route, bestRoute, elemSize);
  
  printf("BEST ROUTE: \n\t");
  
  print_route(bestRoute);
  
  return tmp_route; 
}
/* 
 * A recursive Traveling Salesman Solver using branch-and-bound. 
 * 
 * Returns the shortest roundtrip path based on the starting path described in "route". 
 * 
 * The returned path is "route" if "route" already includes all the bags
 * in the route. If not, route will be freed, and a new path will be returned. 
 * 
 * NB: this function is destructive - it calls free() on route if it finds
 *     a better route than the provided route! 
 * 
 * NB2: there is a slight problem with the below code: ShortestRoute will return a 
 *      semi-intialized bestRoute if all the new permitations are longer than 
 *      globalBest. It shouldn't cause any problems though, as that route 
 *      will be thrown away. 
 */ 
RouteDefinition *ShortestRoute(RouteDefinition *route)
{ 
  RouteDefinition *bestRoute;
  
  int nodesAtThisLevel = nodesAtLevel(fanOutLevel); 

  stack_t* stck;
  stck = stack_create();
	     
  RouteDefinition *newRoute;
      
  float newLength;      
  push(stck, route);
	
  RouteDefinition *curr_route;
  curr_route = Alloc_RouteDefinition();
	
  while(stck->size > 0){
    curr_route = (RouteDefinition *)pop_back(stck);
      
      if (curr_route->nCitiesVisited == fanOutLevel){
	
	// Push current popped node to stack
	push(stck, curr_route);
	
	// convert stack to array
	char *array = stackToArray(stck);
	
	stack_destroy(stck);
	
	char *devArray;
	
	hipMalloc((void**)&devArray, arraySize);
	
	hipMemcpy((void*)devArray, (void*)array, arraySize, hipMemcpyHostToDevice);
	
	float *distArray, *devDistArray;
	
	distArray = flatten_dist_table();
	
	hipMalloc((void**)&devDistArray, dist_array_size);
	
	hipMemcpy((void*)devDistArray, (void*)distArray, dist_array_size, hipMemcpyHostToDevice);
	
	//printf("Arraysize before cuda: %d, nodes: %d\n", arraySize, nodesAtThisLevel);
	
	int threadsPerBlock = 512;
	int blocksPerGrid = (nodesAtThisLevel + threadsPerBlock - 1) / threadsPerBlock;
	
	//printf("Thread Per block: %d, blocksPerGrid: %d, nTotalCities: %d\n", threadsPerBlock, blocksPerGrid, nTotalCities);
	
	cudaSolve<<<blocksPerGrid, threadsPerBlock>>>(devArray, devDistArray, nodesAtThisLevel, elemSize, nTotalCities);
	
	hipMemcpy((void*)array, (void*)devArray, arraySize, hipMemcpyDeviceToHost);
	
	bestRoute = findBestRouteInArray(array);
	
	hipFree(devDistArray);
	hipFree(devArray);
	
	free(distArray);
	free(array);
	
	break;          
      } 
      
      else {
	for (int i = curr_route->nCitiesVisited; i < nTotalCities; i++){
	  
	  newLength = curr_route->length + distanceTable[curr_route->path[curr_route->nCitiesVisited-1]][curr_route->path[i]];
	  
	  newRoute = Alloc_RouteDefinition();  
	  
	  memcpy(newRoute->path, curr_route->path, nTotalCities);   // Copy current route from route
	  
	  // Swaps the position of bag # 'i' and bag # 'nCitiesVisited' from route
	  newRoute->path[curr_route->nCitiesVisited] = curr_route->path[i];
	  newRoute->path[i]              = curr_route->path[curr_route->nCitiesVisited]; 
	  newRoute->nCitiesVisited = curr_route->nCitiesVisited + 1;
	  newRoute->length  = newLength;
		  
	  push(stck, newRoute);
	  
	}   
      } 
    }   

    free(route);

    return bestRoute;
}
	
	// In the desert, the shortest route is a straight line :)
float EuclidDist(Coord *from, Coord *to)
  { 
    float dx = fabs(from->x - to->x);
    float dy = fabs(from->y - to->y);
    return sqrt(dx*dx + dy*dy);
  }


  
// Reads coordinates from a file and generates a distance-table
static void ReadRoute()
  { 
    FILE *file = fopen("./route.dat", "r");
    int i,j;
    
    // Read how many bags there are
    if (fscanf(file, "%d", &nTotalCities) != 1) 
    {
      printf("Error: couldn't read number of bags from route definition file.\n");
      exit(-1);
    }
    
    // Allocate array of bag coords. 
    cityCoords = (Coord*) malloc(nTotalCities * sizeof(Coord)); 
    
    // Read the coordinates of each grain bag
    for (i = 0; i < nTotalCities; i++)
    {
      if (fscanf(file,"%d %d", &cityCoords[i].x, &cityCoords[i].y) != 2) 
      {
	printf("Error: missing or invalid definition of coordinate %d.\n", i);
	exit(-1);
      }
    }
    
    // Allocate distance table 
    distanceTable = (float**) malloc(nTotalCities * sizeof(float*));
    for (i = 0; i < nTotalCities; i++)
      distanceTable[i] = (float*) malloc(nTotalCities * sizeof(float));
    
    // Compute the distances between each of the grain bags.
      for (i = 0; i < nTotalCities; i++)	  
	for (j = 0; j < nTotalCities; j++)	  
	  distanceTable[i][j] = EuclidDist(&cityCoords[i], &cityCoords[j]);
  }
	
	
int main (int argc, char **argv) 
{
  RouteDefinition *originalRoute, *res;
  int i;
  char buf[256];
  
  ReadRoute();
  
  #ifdef GRAPHICS
  gs_init(501,501);
  #endif
  
  // Set up an initial path that goes through each bag in turn. 
  originalRoute = Alloc_RouteDefinition(); 
  for (i = 0; i < nTotalCities; i++)
    originalRoute->path[i] = (unsigned char) i;
  
  #ifdef GRAPHICS
    // Show the original route
    PlotRoute((char *)originalRoute->path); 
    #endif
    
    originalRoute->length = 0.0;
    originalRoute->nCitiesVisited = 1;
    
    sw_init();
    sw_start();
    // Find the best route
    res = ShortestRoute(originalRoute);
    
    sw_stop();
    sw_timeString(buf);
    
    printf("Route length is %lf it took %s\n", res->length, buf);
      
    #ifdef GRAPHICS
      // Show the best route
    PlotRoute((char *)res->path);
      
    sleep(5);
      
    free(res);
      
    gs_exit();
    #endif  
    
    return 0;
    
}
